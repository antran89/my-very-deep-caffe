#include "hip/hip_runtime.h"
#include <functional>
#include <utility>
#include <vector>

#include "caffe/layers/multiview_average_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MultiviewAverageForward(const int nthreads, const Dtype* const bottom_data,
                                        const int channels, const int height,
                                        const int width, const int num_views, Dtype* const top_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int w = index % width;
        const int h = (index / width) % height;
        const int c = (index / width / height) % channels;
        const int n = index / width / height / channels;
        Dtype aveval = 0;
        for (int v = 0; v < num_views; v++) {
            int bottom_index = (((n * num_views + v) * channels + c) * height + h) * width + w;
            aveval += bottom_data[bottom_index];
        }
        top_data[index] = aveval / num_views;
    }
}

template <typename Dtype>
void MultiviewAverageLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                               const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int count = top[0]->count();
    const int num_views = CAFFE_NUM_TEST_VIEWS;
    MultiviewAverageForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
             count, bottom_data, channels_, height_, width_, num_views, top_data);
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(MultiviewAverageLayer);

}  // namespace caffe
