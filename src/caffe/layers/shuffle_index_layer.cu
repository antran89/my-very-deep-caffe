#include "hip/hip_runtime.h"
#include <algorithm>
#include <utility>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "caffe/layers/shuffle_index_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template<typename Dtype>
__global__ void ShuffleIndexForward(const int num_axes, const int* d_bottom_shape, const int* d_top_shape,
                                    const int* d_new_axes, const int bottom_count,
                                    const Dtype* bottom_data, Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, bottom_count) {
        // dynamic array allocation on gpu is too expensive, must roll
        // out a loopy shuffling index computation.
        // This loopy shuffling index implementation is cheap because
        // num_axes is usually small.
        int top_offset = 0, index_cnt, bot_axis, top_index;
        for (int i = 0; i < num_axes; i++) {
            bot_axis = d_new_axes[i];
            index_cnt = index;
            for (int k = num_axes-1; k > bot_axis; k--) {
                index_cnt /= d_bottom_shape[i];
            }
            top_index = index_cnt % d_bottom_shape[i];

            // accumulate top_offset
            top_offset *= d_top_shape[i];
            top_offset += top_index;
        }

        // moving data elements
        top_data[top_offset] = bottom_data[index];
    }
}

template<typename Dtype>
void ShuffleIndexLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom,
                                           const vector<Blob<Dtype> *> &top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int bottom_count = bottom[0]->count();
    vector<int> bottom_shape = bottom[0]->shape();
    vector<int> top_shape = top[0]->shape();
    int num_axes = bottom_shape.size();
    if (bottom_count == 0) return;

    // allocate memory for bottom_shape and top_shape on global GPU memory
    int *d_bottom_shape, *d_top_shape, *d_new_axes;
    hipMalloc(&d_bottom_shape, num_axes);
    hipMemcpy(d_bottom_shape, &bottom_shape[0], num_axes, hipMemcpyHostToDevice);
    hipMalloc(&d_top_shape, num_axes);
    hipMemcpy(d_top_shape, &top_shape[0], num_axes, hipMemcpyHostToDevice);
    hipMalloc(&d_new_axes, num_axes);
    hipMemcpy(d_new_axes, &new_axes_[0], num_axes, hipMemcpyHostToDevice);

    ShuffleIndexForward<Dtype> <<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
    num_axes, d_bottom_shape, d_top_shape, d_new_axes, bottom_count, bottom_data, top_data);
    CUDA_POST_KERNEL_CHECK;
    hipFree(d_bottom_shape);
    hipFree(d_top_shape);
    hipFree(d_new_axes);
}

template<typename Dtype>
__global__ void ShuffleIndexBackward(const int num_axes, const int* d_bottom_shape, const int* d_top_shape,
                                    const int* d_new_axes, const int bottom_count,
                                    const Dtype* top_diff, Dtype* bottom_diff) {
    CUDA_KERNEL_LOOP(index, bottom_count) {
        // dynamic array allocation on gpu is too expensive, must roll
        // out a loopy shuffling index computation.
        // This loopy shuffling index implementation is cheap because
        // num_axes is usually small.
        int top_offset = 0, index_cnt, bot_axis, top_index;
        for (int i = 0; i < num_axes; i++) {
            bot_axis = d_new_axes[i];
            index_cnt = index;
            for (int k = num_axes-1; k > bot_axis; k--) {
                index_cnt /= d_bottom_shape[i];
            }
            top_index = index_cnt % d_bottom_shape[i];

            // accumulate top_offset
            top_offset *= d_top_shape[i];
            top_offset += top_index;
        }

        // moving data elements
        bottom_diff[index] = top_diff[top_offset];
    }
}

template<typename Dtype>
void ShuffleIndexLayer<Dtype>::Backward_gpu(
        const vector<Blob<Dtype> *> &top, const vector<bool> &propagate_down,
        const vector<Blob<Dtype> *> &bottom) {
    if (!propagate_down[0]) {
      return;
    }
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    int bottom_count = bottom[0]->count();
    vector<int> bottom_shape = bottom[0]->shape();
    vector<int> top_shape = top[0]->shape();
    int num_axes = bottom_shape.size();
    if (bottom_count == 0) return;

    // allocate memory for bottom_shape and top_shape on global GPU memory
    int *d_bottom_shape, *d_top_shape, *d_new_axes;
    hipMalloc(&d_bottom_shape, num_axes);
    hipMemcpy(d_bottom_shape, &bottom_shape[0], num_axes, hipMemcpyHostToDevice);
    hipMalloc(&d_top_shape, num_axes);
    hipMemcpy(d_top_shape, &top_shape[0], num_axes, hipMemcpyHostToDevice);
    hipMalloc(&d_new_axes, num_axes);
    hipMemcpy(d_new_axes, &new_axes_[0], num_axes, hipMemcpyHostToDevice);

    ShuffleIndexBackward<Dtype> <<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
    num_axes, d_bottom_shape, d_top_shape, d_new_axes, bottom_count, top_diff, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
    hipFree(d_bottom_shape);
    hipFree(d_top_shape);
    hipFree(d_new_axes);
}

INSTANTIATE_LAYER_GPU_FUNCS(ShuffleIndexLayer);

}  // namespace caffe
